#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include "Imager.h"
#include <stdlib.h> 

using namespace std;

// uncomment to use the camera
//#define CAMERA






int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed
    float gamma = 1.f, sigma = 1.f;
    getParam("g", gamma, argc, argv);
    cout << "gamma: " << gamma << endl;
    getParam("s", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
#if defined(L2) || defined(LAPLACIAN_NORM)
    cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
#else
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
#endif
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    // ### Define your own output images here as needed

    



    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    //float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // gpu vars init
#if defined(GAMMA) || defined(DIVERGENCE)
    int nI = w*h*nc;
    int nO = w*h*mOut.channels();
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *d_imgIn, *d_imgOut;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef L2
    int nI = w*h*nc;
    int nO = w*h;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *d_imgIn, *d_imgOut;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef LAPLACIAN_NORM
    int nI = w*h*nc;
    int nO = w*h;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *d_imgIn, *d_imgOut, *d_gX, *d_gY, *d_divOut;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesI); CUDA_CHECK;
    hipMalloc(&d_divOut, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef CONVOLUTION
    int nI = w*h*nc;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *d_imgIn, *d_imgOut, *k;

    int r = ceil(sigma * 3);
    int d = (2*r)+1;
    k = new float[(size_t)(d * d)];
    kernel(k, r, sigma);
    // cout << "r: " << r << ", d: " << d << ", s: " << sigma << endl;

    // gpu allocs
    /**hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesI); CUDA_CHECK;
    hipMalloc(&d_divOut, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;*/
#endif


    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
#if defined(DIVERGENCE) || defined(L2)
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif
#ifdef LAPLACIAN_NORM
        hipMemset(d_gX, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_gY, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_divOut, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);




    // copy data from host to device
#if defined(GAMMA) || defined(DIVERGENCE) || defined(L2) || defined(LAPLACIAN_NORM)
    hipMemcpy(d_imgIn, imgIn, nbytesI, hipMemcpyHostToDevice); CUDA_CHECK;
#endif
    Timer timer; timer.start();
    // ###
    // ###
    // ### TODO: Main computation
    // ###
    // ###

    for (int i = 0; i < repeats; ++i)
    {
#ifdef GAMMA
    //gamma_correct_host(imgIn, imgOut, w, h, nc, gamma);
    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3(((nI/h) + block.x - 1) / block.x, ((nI/w) + block.y - 1) / block.y, 1);
    gamma_correct_device<<<grid, block>>>(d_imgIn, d_imgOut, gamma, w, h, nI);
#endif
#ifdef DIVERGENCE
    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3(((nI/h) + block.x - 1) / block.x, ((nI/w) + block.y - 1) / block.y, 1);
    divergence<<<grid, block>>>(d_imgIn, d_imgOut, w, h, nI);
#endif
#ifdef L2
    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
    l2_norm<<<grid, block>>>(d_imgIn, d_imgOut, w, h, nc);
#endif
#ifdef LAPLACIAN_NORM
    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3(((nI/h) + block.x - 1) / block.x, ((nI/w) + block.y - 1) / block.y, 1);
    gradient<<<grid, block>>>(d_imgIn, d_gX, d_gY, w, h, nI);
    divergence<<<grid, block>>>(d_gX, d_gY, d_divOut, w, h, nI);
    grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
    l2_norm<<<grid, block>>>(d_divOut, d_imgOut, w, h, nc);
#endif
    }
    hipDeviceSynchronize();  CUDA_CHECK;
    timer.end();  float t = timer.get() / (float) repeats;  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;

    // copy data from device to host
#if defined(GAMMA) || defined(DIVERGENCE) || defined(L2) || defined(LAPLACIAN_NORM)
    hipMemcpy(imgOut, d_imgOut, nbytesO, hipMemcpyDeviceToHost); CUDA_CHECK;
#endif

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif


    // gpu vars deallocs
#if defined(GAMMA) || defined(DIVERGENCE) || defined(L2)
    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
#endif
#ifdef LAPLACIAN_NORM
    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    hipFree(d_gX); CUDA_CHECK;
    hipFree(d_gY); CUDA_CHECK;
    hipFree(d_divOut); CUDA_CHECK;
#endif


    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



