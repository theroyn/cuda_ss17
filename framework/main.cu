#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include "Imager.h"
#include <stdlib.h> 

using namespace std;

// uncomment to use the camera
#define CAMERA






int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed
    float gamma = 1.f, sigma = 1.f, factVal = 1.f, alpha, beta;
    getParam("g", gamma, argc, argv);
    cout << "gamma: " << gamma << endl;
    getParam("s", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;
    getParam("f", factVal, argc, argv);
    cout << "factor: " << factVal << endl;
    getParam("a", alpha, argc, argv);
    cout << "alpha: " << alpha << endl;
    getParam("b", beta, argc, argv);
    cout << "beta: " << beta << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
#if defined(L2) || defined(LAPLACIAN_NORM)
    cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
#else
    cout << "mIn.channels():" << mIn.channels() << endl;
    cout << "mIn.type():" << mIn.type() << endl;
    cout << "CV_32FC1:" << CV_32FC1 << endl;
    cout << "CV_32FC3:" << CV_32FC3 << endl;
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
#endif
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    // ### Define your own output images here as needed
#ifdef STRUCTURE_TENSOR
    cv::Mat mOut11(h,w,CV_32FC1); 
    cv::Mat mOut12(h,w,CV_32FC1); 
    cv::Mat mOut22(h,w,CV_32FC1); 
#endif
    



    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];
    //std::unique_ptr<float[]> imgIn(new float[(size_t)w*h*nc]);

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    //float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // gpu vars init
#ifdef GAMMA
    int nI = w*h*nc;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    //std::unique_ptr<float[]> imgOut(new float[(size_t)nO]);
    float *d_imgIn, *d_imgOut;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef GRADIENT
    int nI = w*h*nc;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    //std::unique_ptr<float[]> imgOut(new float[(size_t)nO]);
    float *d_imgIn, *d_imgOut, *d_gX, *d_gY;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesO); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesO); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef DIVERGENCE
    int nI = w*h*nc;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    //std::unique_ptr<float[]> imgOut(new float[(size_t)nO]);
    float *d_imgIn, *d_imgOut, *d_gX, *d_gY;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef L2
    int nI = w*h*nc;
    int nO = w*h;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    //std::unique_ptr<float[]> imgOut(new float[(size_t)nO]);
    float *d_imgIn, *d_imgOut;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef LAPLACIAN_NORM
    int nI = w*h*nc;
    int nO = w*h;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    //std::unique_ptr<float[]> imgOut(new float[(size_t)nO]);
    float *d_imgIn, *d_imgOut, *d_gX, *d_gY, *d_divOut;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesI); CUDA_CHECK;
    hipMalloc(&d_divOut, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#if defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) || defined(CONVOLUTION_TEXTURE)
    int nI = w*h*nc;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    //std::unique_ptr<float[]> imgOut(new float[(size_t)nO]);
    float *d_imgIn, *d_imgOut, *k, *d_k;
    int r = ceil(sigma * 3);
    int d = (2*r)+1;
    k = new float[(size_t)(d * d)];
    kernel(k, r, sigma);
    cv::Mat mKer(d, d, CV_32FC1);
    float *kt = new float[(size_t)(d * d)];
    scale(k, kt, d*d);
    convert_layered_to_mat(mKer, kt);
    cout << "mKer: " << mKer << endl;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
    int nK = d*d; // kernel
    size_t nbytesK = (size_t)(nK)*sizeof(float);
    hipMalloc(&d_k, nbytesK); CUDA_CHECK;
    hipMemcpy(d_k, k, nbytesK, hipMemcpyHostToDevice); CUDA_CHECK;
#endif
#ifdef CONVOLUTION_CONSTANT
    int nI = w*h*nc;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *d_imgIn, *d_imgOut, *k;

    int r = ceil(sigma * 3);
    int d = (2*r)+1;
    k = new float[(size_t)(d * d)];
    kernel(k, r, sigma);
    cv::Mat mKer(d, d, CV_32FC1);
    float *kt = new float[(size_t)(d * d)];
    scale(k, kt, d*d);
    convert_layered_to_mat(mKer, kt);
    cout << "mKer: " << mKer << endl;

    hipMemcpyToSymbol(HIP_SYMBOL(constKernel), k, (size_t)(d * d)*sizeof(float)); CUDA_CHECK;

    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif
#ifdef STRUCTURE_TENSOR
    int nI = w*h*nc;
    int nO = w*h;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *imgOut11 = (float *) malloc (nbytesO);
    float *imgOut12 = (float *) malloc (nbytesO);
    float *imgOut22 = (float *) malloc (nbytesO);
    float *d_imgIn, *d_s, *k, *d_gX, *d_gY, *d_m11, *d_m12, *d_m22, *d_imgOut11, *d_imgOut12, *d_imgOut22;

    int r = ceil(sigma * 3);
    int d = (2*r)+1;
    k = new float[(size_t)(d * d)];
    kernel(k, r, sigma);
    hipMemcpyToSymbol(HIP_SYMBOL(constKernel), k, (size_t)(d * d)*sizeof(float)); CUDA_CHECK;


    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_s, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesI); CUDA_CHECK;
    hipMalloc(&d_m11, nbytesO); CUDA_CHECK;
    hipMalloc(&d_m12, nbytesO); CUDA_CHECK;
    hipMalloc(&d_m22, nbytesO); CUDA_CHECK;
    hipMalloc(&d_imgOut11, nbytesO); CUDA_CHECK;
    hipMalloc(&d_imgOut12, nbytesO); CUDA_CHECK;
    hipMalloc(&d_imgOut22, nbytesO); CUDA_CHECK;
#endif
#ifdef FEATURE_DETECTION
    int nI = w*h*nc;
    int nG = w*h;
    int nO = w*h*nc;
    size_t nbytesI = (size_t)(nI)*sizeof(float);
    size_t nbytesG = (size_t)(nG)*sizeof(float);
    size_t nbytesO = (size_t)(nO)*sizeof(float);
    float *imgOut = (float *) malloc (nbytesO);
    float *d_imgIn, *d_s, *k, *d_gX, *d_gY, *d_m11, *d_m12,
            *d_m22, *d_imgOut11, *d_imgOut12, *d_imgOut22, *d_imgOut;

    int r = ceil(sigma * 3);
    int d = (2*r)+1;
    k = new float[(size_t)(d * d)];
    kernel(k, r, sigma);
    hipMemcpyToSymbol(HIP_SYMBOL(constKernel), k, (size_t)(d * d)*sizeof(float)); CUDA_CHECK;


    // gpu allocs
    hipMalloc(&d_imgIn, nbytesI); CUDA_CHECK;
    hipMalloc(&d_s, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gX, nbytesI); CUDA_CHECK;
    hipMalloc(&d_gY, nbytesI); CUDA_CHECK;
    hipMalloc(&d_m11, nbytesG); CUDA_CHECK;
    hipMalloc(&d_m12, nbytesG); CUDA_CHECK;
    hipMalloc(&d_m22, nbytesG); CUDA_CHECK;
    hipMalloc(&d_imgOut11, nbytesG); CUDA_CHECK;
    hipMalloc(&d_imgOut12, nbytesG); CUDA_CHECK;
    hipMalloc(&d_imgOut22, nbytesG); CUDA_CHECK;
    hipMalloc(&d_imgOut, nbytesO); CUDA_CHECK;
#endif


    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
        // Get camera image
        camera >> mIn;
        // convert to float representation (opencv loads image values as single bytes by default)
        mIn.convertTo(mIn,CV_32F);
        // convert range of each channel to [0,1] (opencv default is [0,255])
        mIn /= 255.f;
#endif
#ifdef L2
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif
#ifdef GRADIENT
        hipMemset(d_gX, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_gY, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif
#ifdef DIVERGENCE
        hipMemset(d_gX, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_gY, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif
#ifdef LAPLACIAN_NORM
        hipMemset(d_gX, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_gY, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_divOut, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif 
#if defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) || defined(CONVOLUTION_TEXTURE) || defined(CONVOLUTION_CONSTANT)
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif
#ifdef STRUCTURE_TENSOR
        hipMemset(d_s, 0, nbytesI); CUDA_CHECK;
        hipMemset(d_gX, 0, nbytesI); CUDA_CHECK;
        hipMemset(d_gY, 0, nbytesI); CUDA_CHECK;
        hipMemset(d_m11, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_m12, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_m22, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut11, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut12, 0, nbytesO); CUDA_CHECK;
        hipMemset(d_imgOut22, 0, nbytesO); CUDA_CHECK;
#endif
#ifdef FEATURE_DETECTION
        hipMemset(d_s, 0, nbytesI); CUDA_CHECK;
        hipMemset(d_gX, 0, nbytesI); CUDA_CHECK;
        hipMemset(d_gY, 0, nbytesI); CUDA_CHECK;
        hipMemset(d_m11, 0, nbytesG); CUDA_CHECK;
        hipMemset(d_m12, 0, nbytesG); CUDA_CHECK;
        hipMemset(d_m22, 0, nbytesG); CUDA_CHECK;
        hipMemset(d_imgOut11, 0, nbytesG); CUDA_CHECK;
        hipMemset(d_imgOut12, 0, nbytesG); CUDA_CHECK;
        hipMemset(d_imgOut22, 0, nbytesG); CUDA_CHECK;
        hipMemset(d_imgOut, 0, nbytesO); CUDA_CHECK;
#endif

        // Init raw input image array
        // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
        // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
        // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
        convert_mat_to_layered (imgIn, mIn);




        // copy data from host to device
#if defined(GAMMA) || defined(GRADIENT) || defined(DIVERGENCE) || defined(L2) \
|| defined(LAPLACIAN_NORM) || defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) \
|| defined(CONVOLUTION_TEXTURE) || defined(CONVOLUTION_CONSTANT) \
|| defined(STRUCTURE_TENSOR) || defined(FEATURE_DETECTION)
        hipMemcpy(d_imgIn, imgIn, nbytesI, hipMemcpyHostToDevice); CUDA_CHECK;
        //memset(imgOut, 0, nbytesO);
#endif
#ifdef CONVOLUTION_TEXTURE
        texRef.addressMode[0] = hipAddressModeClamp; // clamp x to border
        texRef.addressMode[1] = hipAddressModeClamp; // clamp y to border
        texRef.filterMode = hipFilterModeLinear; // linear interpolation
        texRef.normalized = false; // access as (x+0.5f,y+0.5f), not as ((x+0.5f)/w,(y+0.5f)/h)
        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(NULL, &texRef, d_imgIn, &desc, w, h*nc, w*sizeof(d_imgIn[0])); CUDA_CHECK;
#endif
        Timer timer; timer.start();
        // ###
        // ###
        // ### TODO: Main computation
        // ###
        // ###

        for (int i = 0; i < repeats; ++i)
        {
#ifdef GAMMA
            //gamma_correct_host(imgIn, imgOut, w, h, nc, gamma);
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            gamma_correct_device<<<grid, block>>>(d_imgIn, d_imgOut, gamma, w, h, nc);
#endif
#ifdef GRADIENT
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            gradient<<<grid, block>>>(d_imgIn, d_gX, d_imgOut, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;//d_imgOut, d_gX, d_gY
#endif
#ifdef DIVERGENCE
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            gradient<<<grid, block>>>(d_imgIn, d_gX, d_gY, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;
            divergence<<<grid, block>>>(d_gX, d_gY, d_imgOut, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;
#endif
#ifdef L2
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
            l2_norm<<<grid, block>>>(d_imgIn, d_imgOut, w, h, nc);
#endif
#ifdef LAPLACIAN_NORM
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            gradient<<<grid, block>>>(d_imgIn, d_gX, d_gY, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;
            divergence<<<grid, block>>>(d_gX, d_gY, d_divOut, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;
            grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
            l2_norm<<<grid, block>>>(d_divOut, d_imgOut, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;
#endif
#ifdef CONVOLUTION
            //conv_host(imgIn, imgOut, k, w, h, nc, r);
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            conv_device<<<grid, block>>>(d_imgIn, d_imgOut, d_k, w, h, r);
#endif
#ifdef CONVOLUTION_CONSTANT
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            conv_device_constant<<<grid, block>>>(d_imgIn, d_imgOut, w, h, r);
#endif
#ifdef CONVOLUTION_TEXTURE
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            conv_device_texture<<<grid, block>>>(d_imgOut, d_k, w, h, r);
#endif
#ifdef CONVOLUTION_SHARED
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            int smw = block.x + 2*r;
            int smh = block.y + 2*r;
            size_t smbytes = smw*smh*sizeof(float);
            conv_device_shared<<<grid, block, smbytes>>>(d_imgIn, d_imgOut, d_k, w, h, r, smw, smh);
            hipDeviceSynchronize();  CUDA_CHECK;
#endif
#ifdef STRUCTURE_TENSOR
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            conv_device_constant<<<grid, block>>>(d_imgIn, d_s, w, h, r); // Compute S = G σ ∗ u
            hipDeviceSynchronize();  CUDA_CHECK;
            gradient<<<grid, block>>>(d_s, d_gX, d_gY, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;

            grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
            // d_mxx are grayscale
            pointwise_product<<<grid, block>>>(d_gX, d_gX, d_m11, w, h, nc);
            pointwise_product<<<grid, block>>>(d_gX, d_gY, d_m12, w, h, nc);
            pointwise_product<<<grid, block>>>(d_gY, d_gY, d_m22, w, h, nc);
            
            conv_device_constant<<<grid, block>>>(d_m11, d_imgOut11, w, h, r);
            conv_device_constant<<<grid, block>>>(d_m12, d_imgOut12, w, h, r);
            conv_device_constant<<<grid, block>>>(d_m22, d_imgOut22, w, h, r);
#endif
#ifdef FEATURE_DETECTION
            dim3 block = dim3(32, 8, 1);
            dim3 grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            conv_device_constant<<<grid, block>>>(d_imgIn, d_s, w, h, r); // Compute S = G σ ∗ u
            hipDeviceSynchronize();  CUDA_CHECK;
            gradient<<<grid, block>>>(d_s, d_gX, d_gY, w, h, nc);
            hipDeviceSynchronize();  CUDA_CHECK;
            /**hipMemcpy(imgOut, d_gX, nbytesO, hipMemcpyDeviceToHost); CUDA_CHECK;
            convert_layered_to_mat(mOut, imgOut);
            cout << "mOutX" << mOut(cv::Range(0,15), cv::Range(0,15)) << endl;
            showSizeableImage("OutputX", mOut, 100+w+40, 100);*/

            grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
            // d_mxx are grayscale
            pointwise_product<<<grid, block>>>(d_gX, d_gX, d_m11, w, h, nc);
            pointwise_product<<<grid, block>>>(d_gX, d_gY, d_m12, w, h, nc);
            pointwise_product<<<grid, block>>>(d_gY, d_gY, d_m22, w, h, nc);
            
            conv_device_constant<<<grid, block>>>(d_m11, d_imgOut11, w, h, r);
            conv_device_constant<<<grid, block>>>(d_m12, d_imgOut12, w, h, r);
            conv_device_constant<<<grid, block>>>(d_m22, d_imgOut22, w, h, r);
            hipDeviceSynchronize();  CUDA_CHECK;

            //grid = dim3((w + block.x - 1) / block.x, (h*nc + block.y - 1) / block.y, 1);
            feature_detect<<<grid, block>>>(d_imgIn, d_imgOut11, d_imgOut12, d_imgOut22, d_imgOut, w, h, alpha, beta);
            hipDeviceSynchronize();  CUDA_CHECK;
#endif
        }
        hipDeviceSynchronize();  CUDA_CHECK;
        timer.end();  float t = timer.get() / (float) repeats;  // elapsed time in seconds
        cout << "time: " << t*1000 << " ms" << endl;

        // copy data from device to host
#if defined(GAMMA) || defined(GRADIENT) || defined(DIVERGENCE) || \
defined(L2) || defined(LAPLACIAN_NORM) || defined(CONVOLUTION) \
|| defined(CONVOLUTION_SHARED) || defined(CONVOLUTION_TEXTURE) \
|| defined(CONVOLUTION_CONSTANT) || defined(FEATURE_DETECTION)
        hipMemcpy(imgOut, d_imgOut, nbytesO, hipMemcpyDeviceToHost); CUDA_CHECK;
#endif
#ifdef STRUCTURE_TENSOR
        hipMemcpy(imgOut11, d_imgOut11, nbytesO, hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(imgOut12, d_imgOut12, nbytesO, hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(imgOut22, d_imgOut22, nbytesO, hipMemcpyDeviceToHost); CUDA_CHECK;
#endif

        // show input image
        showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

#ifndef STRUCTURE_TENSOR
        // show output image: first convert to interleaved opencv format from the layered raw array
        convert_layered_to_mat(mOut, imgOut);
        showImage("Output", mOut, 100+w+40, 100);
#else
        convert_layered_to_mat(mOut11, imgOut11);
        convert_layered_to_mat(mOut12, imgOut12);
        convert_layered_to_mat(mOut22, imgOut22);
        mOut11 *= factVal;
        mOut12 *= factVal;
        mOut22 *= factVal;
        cout << "mOut11" << mOut11(cv::Range(0,5), cv::Range(0,5)) << endl;
        showImage("Output11", mOut11, 100+w+40, 100);
        showImage("Output12", mOut12, 100, 100+h+40);
        showImage("Output22", mOut22, 100+w+40, 100+h+40);
#endif

        // ### Display your own output images here as needed
#if defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) || defined(CONVOLUTION_TEXTURE) || defined(CONVOLUTION_CONSTANT)
        showSizeableImage("Kernel", mKer, 100, 100+h+40);
#endif

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

//all
//imgIn X
//ifdef GAMMA
//imgOut, d_imgIn, *d_imgOut
//ifdef GRADIENT
//imgOut, d_imgIn, *d_imgOut, *d_gX, *d_gY
//ifdef DIVERGENCE
//imgOut, d_imgIn, *d_imgOut, *d_gX, *d_gY;
//ifdef L2
//imgOut, d_imgIn, *d_imgOut
//ifdef LAPLACIAN_NORM
//imgOut, d_imgIn, *d_imgOut, *d_gX, *d_gY, *d_divOut
//if defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) || defined(CONVOLUTION_TEXTURE)
//imgOut, *d_imgIn, *d_imgOut, *k, *d_k, kt
//ifdef CONVOLUTION_CONSTANT
//imgOut, *d_imgIn, *d_imgOut, *k, kt
    hipDeviceSynchronize();  CUDA_CHECK;
    //cpu deallocs
    free(imgIn);
    delete[] imgOut;
#if defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) \
|| defined(CONVOLUTION_TEXTURE) || defined(CONVOLUTION_CONSTANT)
    delete[] k;
    delete[] kt;
#endif
    // gpu vars deallocs
#if defined(GAMMA) || defined(L2) || defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) \
|| defined(CONVOLUTION_TEXTURE) || defined(CONVOLUTION_CONSTANT)
    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
#endif
#ifdef CONVOLUTION_TEXTURE
    hipUnbindTexture(texRef);
#endif
#ifdef GRADIENT
    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    hipFree(d_gX); CUDA_CHECK;
    hipFree(d_gY); CUDA_CHECK;
#endif
#ifdef DIVERGENCE
    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    hipFree(d_gX); CUDA_CHECK;
    hipFree(d_gY); CUDA_CHECK;
#endif
#ifdef LAPLACIAN_NORM
    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
    hipFree(d_gX); CUDA_CHECK;
    hipFree(d_gY); CUDA_CHECK;
    hipFree(d_divOut); CUDA_CHECK;
#endif
#if defined(CONVOLUTION) || defined(CONVOLUTION_SHARED) || defined(CONVOLUTION_TEXTURE)
    hipFree(d_k); CUDA_CHECK;
#endif
#ifdef STRUCTURE_TENSOR
// d_gX, *d_gY, *d_m11, *d_m12, *d_m22, *d_imgOut11, *d_imgOut12, *d_imgOut22

    cout << "nbytesI: " << nbytesI << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_gX); CUDA_CHECK;
    hipFree(d_gY); CUDA_CHECK;
    hipFree(d_m11); CUDA_CHECK;
    hipFree(d_m12); CUDA_CHECK;
    hipFree(d_m22); CUDA_CHECK;
    hipFree(d_imgOut11); CUDA_CHECK;
    hipFree(d_imgOut12); CUDA_CHECK;
    hipFree(d_imgOut22); CUDA_CHECK;

    
    free(imgOut11);
    free(imgOut12);
    free(imgOut22);
#endif
#ifdef FEATURE_DETECTION
// d_gX, *d_gY, *d_m11, *d_m12, *d_m22, *d_imgOut11, *d_imgOut12, *d_imgOut22

    cout << "nbytesI: " << nbytesI << " nbytesG: " << nbytesG << " nbytesO: " << nbytesO << endl;
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_gX); CUDA_CHECK;
    hipFree(d_gY); CUDA_CHECK;
    hipFree(d_m11); CUDA_CHECK;
    hipFree(d_m12); CUDA_CHECK;
    hipFree(d_m22); CUDA_CHECK;
    hipFree(d_imgOut11); CUDA_CHECK;
    hipFree(d_imgOut12); CUDA_CHECK;
    hipFree(d_imgOut22); CUDA_CHECK;
    hipFree(d_imgOut); CUDA_CHECK;
#endif


    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    //delete[] imgIn;
    //delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



