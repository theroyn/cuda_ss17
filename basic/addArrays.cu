// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}


__device__ void add_array(float *a, float *b, float *c, int n)
{
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx<n) c[idx] = a[idx] + b[idx];
}

__global__ void add_array_wrapper(float *a, float *b, float *c, int n)
{
    add_array(a, b, c, n);
}

int main(int argc, char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 20;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];
    for(int i=0; i<n; i++)
    {
        a[i] = i;
        b[i] = (i%5)+1;
        c[i] = 0;
    }

    // CPU computation
    for(int i=0; i<n; i++) c[i] = a[i] + b[i];

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;
    // init c
    for(int i=0; i<n; i++) c[i] = 0;
    
    // copy to device
    float *d_a, *d_b, *d_c;
    size_t nbytes = (size_t)(n)*sizeof(int);
    hipMalloc(&d_a, nbytes); CUDA_CHECK;
    hipMalloc(&d_b, nbytes); CUDA_CHECK;
    hipMalloc(&d_c, nbytes); CUDA_CHECK;
    hipMemcpy(d_a, a, nbytes, hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_b, b, nbytes, hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemcpy(d_c, c, nbytes, hipMemcpyHostToDevice); CUDA_CHECK;
    
    // launch kernel
    dim3 block = dim3(128,1,1);
    // dim3 grid = dim3((n + block.x –1) / block.x, 1, 1);
    dim3 grid = dim3((n+block.x-1)/block.x,1,1);

    add_array_wrapper<<<grid, block>>>(d_a, d_b, d_c, n);

    // copy to host and deallocate
    hipMemcpy(c, d_c, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
    hipFree(d_a); CUDA_CHECK;
    hipFree(d_b); CUDA_CHECK;
    hipFree(d_c); CUDA_CHECK;


    // GPU computation
    // ###
    // ### TODO: Implement the array addition on the GPU, store the result in "c"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "helper.h"
    


    // print result
    cout << "GPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
    delete[] b;
    delete[] c;
}



