// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}




int main(int argc,char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 10;
    float *a = new float[n];
    for(int i=0; i<n; i++) a[i] = i;

    // CPU computation
    for(int i=0; i<n; i++)
    {
        float val = a[i];
        val = val*val;
        a[i] = val;
    }

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;
    


    // GPU computation
    // reinit data
    for(int i=0; i<n; i++) a[i] = i;

    
    // ###
    // ### TODO: Implement the "square array" operation on the GPU and store the result in "a"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "helper.h"


    // print result
    cout << "GPU:" << endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
}



